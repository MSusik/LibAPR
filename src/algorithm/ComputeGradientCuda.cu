#include "hip/hip_runtime.h"
#include "ComputeGradientCuda.hpp"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>


__global__ void gradient(float *input, size_t x_num, size_t y_num, size_t z_num, float *grad, size_t x_num_ds, size_t y_num_ds, float hx, float hy, float hz) {
    int xi = ((blockIdx.x * blockDim.x) + threadIdx.x) * 2;
    int yi = ((blockIdx.y * blockDim.y) + threadIdx.y) * 2;
    int zi = ((blockIdx.z * blockDim.z) + threadIdx.z) * 2;
    if (xi >= x_num || yi >= y_num || zi >= z_num) return;

    const size_t xnumynum = x_num * y_num;

    float temp[4][4][4];

    for (int z = 0; z < 4; ++z)
        for (int x = 0; x < 4; ++x)
            for(int y = 0; y < 4; ++y) {
                int xc = xi + x - 1; if (xc < 0) xc = 0; else if (xc >= x_num) xc = x_num - 1;
                int yc = yi + y - 1; if (yc < 0) yc = 0; else if (yc >= y_num) yc = y_num - 1;
                int zc = zi + z - 1; if (zc < 0) zc = 0; else if (zc >= z_num) zc = z_num - 1;
                temp[z][x][y] = *(input + zc * xnumynum + xc * y_num + yc);
            }
    float maxGrad = 0;
    for (int z = 1; z <= 2; ++z)
        for (int x = 1; x <= 2; ++x)
            for(int y = 1; y <= 2; ++y) {
                float xd = (temp[z][x-1][y] - temp[z][x+1][y]) / (2 * hx); xd = xd * xd;
                float yd = (temp[z-1][x][y] - temp[z+1][x][y]) / (2 * hy); yd = yd * yd;
                float zd = (temp[z][x][y-1] - temp[z][x][y+1]) / (2 * hz); zd = zd * zd;
                float gm = __fsqrt_rn(xd + yd + zd);
                if (gm > maxGrad)  maxGrad = gm;
            }

    const size_t idx = zi/2 * x_num_ds * y_num_ds + xi/2 * y_num_ds + yi/2;
    grad[idx] = maxGrad;
}

void cudaDownsampledGradient(const MeshData<float> &input, MeshData<float> &grad, const float hx, const float hy,const float hz) {
    APRTimer timer;
    timer.verbose_flag=true;

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t inputSize = input.mesh.size() * sizeof(float);
    float *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    size_t gradSize = grad.mesh.size() * sizeof(float);
    float *cudaGrad;
    hipMalloc(&cudaGrad, gradSize);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(4, 4, 4);
    dim3 numBlocks((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   (input.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   (input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
    std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;

    gradient<<<numBlocks,threadsPerBlock>>>(cudaInput, input.x_num, input.y_num, input.z_num, cudaGrad, grad.x_num, grad.y_num, hx, hy, hz);
    hipDeviceSynchronize();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy((void*)input.mesh.get(), cudaInput, inputSize, hipMemcpyDeviceToHost);
    hipFree(cudaInput);
    hipMemcpy((void*)grad.mesh.get(), cudaGrad, gradSize, hipMemcpyDeviceToHost);
    hipFree(cudaGrad);
    timer.stop_timer();
}

/////////////////////////////////////////////////////////////////////


float impulse_resp(float k,float rho,float omg){
    //  Impulse Response Function
    return (pow(rho,(std::abs(k)))*sin((std::abs(k) + 1)*omg)) / sin(omg);
}

float impulse_resp_back(float k,float rho,float omg,float gamma,float c0){
    //  Impulse Response Function (nominator eq. 4.8, denominator from eq. 4.7)
    return c0*pow(rho,std::abs(k))*(cos(omg*std::abs(k)) + gamma*sin(omg*std::abs(k)))*(1.0/(pow((1 - 2.0*rho*cos(omg) + pow(rho,2)),2)));
}


typedef struct {
    std::vector<float> bc1_vec;
    std::vector<float> bc2_vec;
    std::vector<float> bc3_vec;
    std::vector<float> bc4_vec;
    size_t k0;
    float b1;
    float b2;
    float norm_factor;
} BsplineParams;

BsplineParams prepareBsplineStuff(MeshData<float> & image, float lambda, float tol) {
    float xi = 1 - 96*lambda + 24*lambda*sqrt(3 + 144*lambda); // eq 4.6
    float rho = (24*lambda - 1 - sqrt(xi))/(24*lambda)*sqrt((1/xi)*(48*lambda + 24*lambda*sqrt(3 + 144*lambda))); // eq 4.5
    float omg = atan(sqrt((1/xi)*(144*lambda - 1))); // eq 4.6

    float c0 = (1+ pow(rho,2))/(1-pow(rho,2)) * (1 - 2*rho*cos(omg) + pow(rho,2))/(1 + 2*rho*cos(omg) + pow(rho,2)); // eq 4.8
    float gamma = (1-pow(rho,2))/(1+pow(rho,2)) * (1/tan(omg)); // eq 4.8

    const float b1 = 2*rho*cos(omg);
    const float b2 = -pow(rho,2.0);

    const size_t z_num = image.z_num;

    const size_t k0 = std::max(std::min((size_t)(ceil(std::abs(log(tol)/log(rho)))),z_num),(size_t)2);
    const float norm_factor = pow((1 - 2.0*rho*cos(omg) + pow(rho,2)),2);

    //////////////////////////////////////////////////////////////
    //
    //  Setting up boundary conditions
    //
    //////////////////////////////////////////////////////////////

    // for boundaries
    std::cout << "k0=" << k0 << std::endl;
    std::vector<float> impulse_resp_vec_f(k0+3);  //forward
    for (size_t k = 0; k < (k0+3); ++k) {
        impulse_resp_vec_f[k] = impulse_resp(k,rho,omg);
    }

    std::vector<float> impulse_resp_vec_b(k0+3);  //backward
    for (size_t k = 0; k < (k0+3); ++k) {
        impulse_resp_vec_b[k] = impulse_resp_back(k,rho,omg,gamma,c0);
    }

    std::vector<float> bc1_vec(k0, 0);  //forward
    //y(1) init
    bc1_vec[1] = impulse_resp_vec_f[0];
    for (size_t k = 0; k < k0; ++k) {
        bc1_vec[k] += impulse_resp_vec_f[k+1];
    }

    std::vector<float> bc2_vec(k0, 0);  //backward
    //y(0) init
    for (size_t k = 0; k < k0; ++k) {
        bc2_vec[k] = impulse_resp_vec_f[k];
    }

    std::vector<float> bc3_vec(k0, 0);  //forward
    //y(N-1) init
    bc3_vec[0] = impulse_resp_vec_b[1];
    for (size_t k = 0; k < (k0-1); ++k) {
        bc3_vec[k+1] += impulse_resp_vec_b[k] + impulse_resp_vec_b[k+2];
    }

    std::vector<float> bc4_vec(k0, 0);  //backward
    //y(N) init
    bc4_vec[0] = impulse_resp_vec_b[0];
    for (size_t k = 1; k < k0; ++k) {
        bc4_vec[k] += 2*impulse_resp_vec_b[k];
    }

    return BsplineParams {
            bc1_vec,
            bc2_vec,
            bc3_vec,
            bc4_vec,
            k0,
            b1,
            b2,
            norm_factor
    };
}

//
//__global__ void bsplineY(float *image, size_t x_num, size_t y_num, size_t z_num, float *bc1_vec, float *bc2_vec, float *bc3_vec, float *bc4_vec, size_t k0, float b1, float b2, float norm_factor) {
//    int xi = ((blockIdx.x * blockDim.x) + threadIdx.x);
//    int zi = ((blockIdx.z * blockDim.z) + threadIdx.z);
//    __shared__ float bc1_vec2[20];
//    __shared__ float bc2_vec2[20];
//    __shared__ float bc3_vec2[20];
//    __shared__ float bc4_vec2[20];
//    uint idx = blockDim.x * threadIdx.z + threadIdx.x;
//    if (idx < 4) {
//        if (idx == 0) for (int i = 0; i < k0; ++i) bc1_vec2[i] = bc1_vec[i];
//        else if (idx == 1) for (int i = 0; i < k0; ++i) bc2_vec2[i] = bc2_vec[i];
//        else if (idx == 2) for (int i = 0; i < k0; ++i) bc3_vec2[i] = bc3_vec[i];
//        else if (idx == 3) for (int i = 0; i < k0; ++i) bc4_vec2[i] = bc4_vec[i];
//    }
//    __syncthreads();
//
//    //forwards direction
//    const size_t zPlaneOffset = zi * x_num * y_num;
//    const size_t yColOffset = xi * y_num;
//    size_t yCol = zPlaneOffset + yColOffset;
//
//    float temp1 = 0;
//    float temp2 = 0;
//    float temp3 = 0;
//    float temp4 = 0;
//
//    for (size_t k = 0; k < k0; ++k) {
//        temp1 += bc1_vec2[k]*image[yCol + k];
//        temp2 += bc2_vec2[k]*image[yCol + k];
//        temp3 += bc3_vec2[k]*image[yCol + y_num - 1 - k];
//        temp4 += bc4_vec2[k]*image[yCol + y_num - 1 - k];
//    }
//
//    //initialize the sequence
//    image[yCol + 0] = temp2;
//    image[yCol + 1] = temp1;
//
//    // middle values
//    for (auto it = (image + yCol + 2); it !=  (image+yCol + y_num); ++it) {
//        float  temp = temp1*b1 + temp2*b2 + *it;
//        *it = temp;
//        temp2 = temp1;
//        temp1 = temp;
//    }
//
//    // finish sequence
//    image[yCol + y_num - 2] = temp3;
//    image[yCol + y_num - 1] = temp4;
//
//    // -------------- part 2
//    temp2 = image[yCol + y_num - 1];
//    temp1 = image[yCol + y_num - 2];
//    image[yCol + y_num - 1]*=norm_factor;
//    image[yCol + y_num - 2]*=norm_factor;
//
//    for (auto it = (image + yCol + y_num-3); it !=  (image + yCol - 1); --it) {
//        float temp = temp1*b1 + temp2*b2 + *it;
//        *it = temp*norm_factor;
//        temp2 = temp1;
//        temp1 = temp;
//    }
//
//}


__global__ void bsplineY(float *image, size_t x_num, size_t y_num, size_t z_num, float *bc1_vec, float *bc2_vec, float *bc3_vec, float *bc4_vec, size_t k0, float b1, float b2, float norm_factor) {
    int xi = ((blockIdx.x * blockDim.x) + threadIdx.x);
    int zi = ((blockIdx.z * blockDim.z) + threadIdx.z);

    __shared__ float bc1_vec2[32];
    __shared__ float bc2_vec2[32];
    __shared__ float bc3_vec2[32];
    __shared__ float bc4_vec2[32];

    uint idx = blockDim.x * threadIdx.z + threadIdx.x;
    if (idx == 0) for (int i = 0; i < k0; ++i) bc1_vec2[i] = bc1_vec[i];
    else if (idx == 1) for (int i = 0; i < k0; ++i) bc2_vec2[i] = bc2_vec[i];
    else if (idx == 2) for (int i = 0; i < k0; ++i) bc3_vec2[i] = bc3_vec[i];
    else if (idx == 3) for (int i = 0; i < k0; ++i) bc4_vec2[i] = bc4_vec[i];

    __syncthreads();

    //forwards direction
    const size_t zPlaneOffset = zi * x_num * y_num;
    const size_t yColOffset = xi * y_num;
    const size_t yCol = zPlaneOffset + yColOffset;

    float cache[1024];

    float temp1 = 0;
    float temp2 = 0;
    for (size_t k = 0; k < k0; ++k) {
        temp1 += bc1_vec2[k] * image[yCol + k];
        temp2 += bc2_vec2[k] * image[yCol + k];
    }
    float temp3 = 0;
    float temp4 = 0;
    for (size_t k = 0; k < k0; ++k) {
        temp3 += bc3_vec2[k]*image[yCol + y_num - 1 - k];
        temp4 += bc4_vec2[k]*image[yCol + y_num - 1 - k];
    }

    //initialize the sequence
    cache[0] = temp2;
    cache[1] = temp1;

    // middle values
    float *it = image + yCol + 2;
    for (int i = 2; i < y_num; ++i, ++it) {
        float  temp = temp1*b1 + temp2*b2 + *it;
        temp2 = temp1;
        cache[i] = temp;
        temp1 = temp;
    }

    // -------------- part 2

    temp2 = temp4;
    temp1 = temp3;
    image[yCol + y_num - 1] = temp2 * norm_factor;
    image[yCol + y_num - 2] = temp1 * norm_factor;

    it = image + yCol + y_num - 3;
    for (int i = y_num - 3; i >= 0; --i, --it) {
        float temp = temp1*b1 + temp2*b2 + cache[i];
        temp2 = temp1;
        *it = temp*norm_factor;
        temp1 = temp;
    }
}

void cudaFilterBsplineYdirection(MeshData<float> &input, float lambda, float tolerance) {
    APRTimer timer;
    timer.verbose_flag=true;

    timer.start_timer("bspline...");
    BsplineParams p = prepareBsplineStuff(input, lambda, tolerance);
    timer.stop_timer();
    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t inputSize = input.mesh.size() * sizeof(float);
    float *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    thrust::device_vector<float> d_bc1_vec(p.bc1_vec);
    thrust::device_vector<float> d_bc2_vec(p.bc2_vec);
    thrust::device_vector<float> d_bc3_vec(p.bc3_vec);
    thrust::device_vector<float> d_bc4_vec(p.bc4_vec);
    timer.stop_timer();

    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   1,
                   (input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
    std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;

    float *bc1 = thrust::raw_pointer_cast(d_bc1_vec.data());
    float *bc2 = thrust::raw_pointer_cast(d_bc2_vec.data());
    float *bc3 = thrust::raw_pointer_cast(d_bc3_vec.data());
    float *bc4 = thrust::raw_pointer_cast(d_bc4_vec.data());
    bsplineY<<<numBlocks,threadsPerBlock>>>(cudaInput, input.x_num, input.y_num, input.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor);
    hipDeviceSynchronize();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)printf("Error: %s\n", hipGetErrorString(err));

    hipMemcpy((void*)input.mesh.get(), cudaInput, inputSize, hipMemcpyDeviceToHost);
    hipFree(cudaInput);
    timer.stop_timer();
}